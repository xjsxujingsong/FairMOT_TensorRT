#include "hip/hip_runtime.h"


#include "TestPlugin.hpp"

typedef TRTInfer::halfloat halfloat;

template<typename _T>
__global__ void MyPluginKenel(_T* input, _T* output, int edge);

template<>
__global__ void MyPluginKenel(float* input, float* output, int edge) {

	KERNEL_POSITION;
	output[position] = (input[position] < 0 ? 0 : input[position]) + 1.3f;
}

template<>
__global__ void MyPluginKenel(halfloat* input, halfloat* output, int edge) {

	KERNEL_POSITION;

	halfloat zero = 0.0f;
	halfloat add = 1.3f;
	output[position] = (input[position] < zero ? zero : input[position]) + add;
}

nvinfer1::Dims TestPlugin::outputDims(int index, const nvinfer1::Dims* inputDims, int nbInputDims) {
	return inputDims[0];
}

std::shared_ptr<LayerConfig> TestPlugin::config(const std::string& layerName) {
	auto cfg = TRTPlugin::config(layerName);

	//定义我们这个插件支持half和float格式
	cfg->supportDataType_ = {nvinfer1::DataType::kHALF, nvinfer1::DataType::kFLOAT};
	//cfg->supportDataType_ = {nvinfer1::DataType::kHALF};
	return cfg;
}

int TestPlugin::enqueue(const std::vector<Plugin::GTensor>& inputs, std::vector<Plugin::GTensor>& outputs, const std::vector<GTensor>& weights, void* workspace, hipStream_t stream) {

	int count = inputs[0].count();
	auto grid = gridDims(count);
	auto block = blockDims(count);

	if (config_->configDataType_ == TRTInfer::DataType::dtFloat) {
		MyPluginKenel <<<grid, block >>> (inputs[0].ptr<float>(), outputs[0].ptr<float>(), count);
	}
	else if (config_->configDataType_ == TRTInfer::DataType::dtHalfloat) {
		MyPluginKenel <<<grid, block>>> (inputs[0].ptr<halfloat>(), outputs[0].ptr<halfloat>(), count);
	}
	return 0;
}

RegisterPlugin(TestPlugin);