#include "hip/hip_runtime.h"


#include "DCNv2.hpp"
#include <common/json.hpp>
#include <hipblas.h>

typedef TRTInfer::halfloat halfloat;

#define cublasCheck(op)														 \
do {																	 \
	auto ret = (op);													 \
	if (ret != HIPBLAS_STATUS_SUCCESS) {											 \
		INFO("%s fail, %d != %d", #op, ret, HIPBLAS_STATUS_SUCCESS);				 \
		abort();													     \
	}																	 \
} while (0);


template<typename _T>
static __global__ void sigmoidKernel(_T* input, _T* output, int edge);

template<>
__global__ void sigmoidKernel(float* input, float* output, int edge) {

	KERNEL_POSITION;
	output[position] = 1 / (1 + exp(-input[position]));
}

template<>
__global__ void sigmoidKernel(halfloat* input, halfloat* output, int edge) {

	KERNEL_POSITION;
	halfloat one = 1.0f;
	output[position] = one / (one + hexp(-input[position]));
}

static __device__ float dmcnIm2colBilinearFP32(const float *bottom_data, const int data_width,
	const int height, const int width, float h, float w)
{
	int h_low = floor(h);
	int w_low = floor(w);
	int h_high = h_low + 1;
	int w_high = w_low + 1;

	float lh = h - h_low;
	float lw = w - w_low;
	float hh = 1 - lh, hw = 1 - lw;

	float v1 = 0;
	if (h_low >= 0 && w_low >= 0)
		v1 = bottom_data[h_low * data_width + w_low];
	float v2 = 0;
	if (h_low >= 0 && w_high <= width - 1)
		v2 = bottom_data[h_low * data_width + w_high];
	float v3 = 0;
	if (h_high <= height - 1 && w_low >= 0)
		v3 = bottom_data[h_high * data_width + w_low];
	float v4 = 0;
	if (h_high <= height - 1 && w_high <= width - 1)
		v4 = bottom_data[h_high * data_width + w_high];

	float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

	float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
	return val;
}

static __device__ halfloat dmcnIm2colBilinearFP16(const halfloat *bottom_data, const int data_width,
	const int height, const int width, const halfloat& h, const halfloat& w)
{
	int h_low = hfloor(h);
	int w_low = hfloor(w);
	int h_high = h_low + 1;
	int w_high = w_low + 1;

	halfloat one = 1.0f;
	halfloat h_low_hf = h_low;
	halfloat w_low_hf = w_low;
	halfloat lh = h - h_low_hf;
	halfloat lw = w - w_low_hf;
	halfloat hh = one - lh, hw = one - lw;

	halfloat zero = 0.0f;
	halfloat v1 = zero;
	if (h_low >= 0 && w_low >= 0)
		v1 = bottom_data[h_low * data_width + w_low];
	halfloat v2 = zero;
	if (h_low >= 0 && w_high <= width - 1)
		v2 = bottom_data[h_low * data_width + w_high];
	halfloat v3 = zero;
	if (h_high <= height - 1 && w_low >= 0)
		v3 = bottom_data[h_high * data_width + w_low];
	halfloat v4 = zero;
	if (h_high <= height - 1 && w_high <= width - 1)
		v4 = bottom_data[h_high * data_width + w_high];

	halfloat w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
	return (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
}

template<typename _T>
static __global__ void DCNIm2colKernel(
	const _T *data_input, const _T *data_offset, const _T *data_mask,
	const int height_input, const int width_input, const int kernel_h, const int kernel_w,
	const int pad_h, const int pad_w,
	const int stride_h, const int stride_w,
	const int dilation_h, const int dilation_w,
	const int channel_per_deformable_group,
	const int batch_size, const int num_channels, const int deformable_group,
	const int height_output, const int width_output,
	_T *data_output, int edge);

template<>
__global__ void DCNIm2colKernel(
	const float *data_input, const float *data_offset, const float *data_mask,
	const int height_input, const int width_input, const int kernel_h, const int kernel_w,
	const int pad_h, const int pad_w,
	const int stride_h, const int stride_w,
	const int dilation_h, const int dilation_w,
	const int channel_per_deformable_group,
	const int batch_size, const int num_channels, const int deformable_group,
	const int height_output, const int width_output,
	float *data_output, int edge)
{
	KERNEL_POSITION;

	const int f_area_input = width_input * height_input;
	const int f_area_output = width_output * height_output;

	// index index of output matrix
	const int w_output = position % width_output;
	const int h_output = (position / width_output) % height_output;
	const int c_input = (position / width_output / height_output) % num_channels;

	const int c_output = c_input * kernel_h * kernel_w;
	const int deformable_group_index = c_input / channel_per_deformable_group;
	const int h_input = h_output * stride_h - pad_h;
	const int w_input = w_output * stride_w - pad_w;


	int data_output_offset = c_input * kernel_h * kernel_w * f_area_output + h_output * width_output + w_output;
	float *data_output_ptr = data_output + data_output_offset;
	const float *data_input_ptr = data_input + c_input * f_area_input;
	const float *data_offset_ptr = data_offset + deformable_group_index * 2 * kernel_h * kernel_w * f_area_output;
	const float *data_mask_ptr = data_mask + deformable_group_index * kernel_h * kernel_w * f_area_output;

	for (int i = 0; i < kernel_h; ++i)
	{
		for (int j = 0; j < kernel_w; ++j)
		{
			const int row = i + h_input;
			const int col = j + w_input;
			const int kernel_index = i * kernel_w + j;

			const int offset_h_offset = 2 * kernel_index * f_area_output + h_output * width_output + w_output;
			const int offset_w_offset = (2 * kernel_index + 1) * f_area_output + h_output * width_output + w_output;
			const int mask_offset = kernel_index * f_area_output + h_output * width_output + w_output;

			const float offset_h = data_offset_ptr[offset_h_offset];
			const float offset_w = data_offset_ptr[offset_w_offset];
			const float mask = data_mask_ptr[mask_offset];

			float val = 0;
			const float h_im = h_input + i * dilation_h + offset_h;
			const float w_im = w_input + j * dilation_w + offset_w;

			if (h_im > -1 && w_im > -1 && h_im < height_input && w_im < width_input)
			{
				val = dmcnIm2colBilinearFP32(data_input_ptr, width_input, height_input, width_input, h_im, w_im);
			}
			*data_output_ptr = val * mask;
			data_output_ptr += f_area_output;
		}
	}
}

template<>
__global__ void DCNIm2colKernel(
	const halfloat *data_input, const halfloat *data_offset, const halfloat *data_mask,
	const int height_input, const int width_input, const int kernel_h, const int kernel_w,
	const int pad_h, const int pad_w,
	const int stride_h, const int stride_w,
	const int dilation_h, const int dilation_w,
	const int channel_per_deformable_group,
	const int batch_size, const int num_channels, const int deformable_group,
	const int height_output, const int width_output,
	halfloat *data_output, int edge)
{
	KERNEL_POSITION;

	const int f_area_input = width_input * height_input;
	const int f_area_output = width_output * height_output;

	// index index of output matrix
	const int w_output = position % width_output;
	const int h_output = (position / width_output) % height_output;
	const int c_input = (position / width_output / height_output) % num_channels;

	const int c_output = c_input * kernel_h * kernel_w;
	const int deformable_group_index = c_input / channel_per_deformable_group;
	const int h_input = h_output * stride_h - pad_h;
	const int w_input = w_output * stride_w - pad_w;

	halfloat width_input_hf = __float2half(width_input);
	halfloat height_input_hf = __float2half(height_input);

	halfloat h_input_hf = __float2half(h_input);
	halfloat w_input_hf = __float2half(w_input);
	halfloat dilation_h_hf = __float2half(dilation_h);
	halfloat dilation_w_hf = __float2half(dilation_w);

	int data_output_offset = c_input * kernel_h * kernel_w * f_area_output + h_output * width_output + w_output;
	halfloat *data_output_ptr = data_output + data_output_offset;
	const halfloat *data_input_ptr = data_input + c_input * f_area_input;
	const halfloat *data_offset_ptr = data_offset + deformable_group_index * 2 * kernel_h * kernel_w * f_area_output;
	const halfloat *data_mask_ptr = data_mask + deformable_group_index * kernel_h * kernel_w * f_area_output;

	halfloat n_one = -1.0f;
	halfloat zero = 0.0f;

	for (int i = 0; i < kernel_h; ++i)
	{
		for (int j = 0; j < kernel_w; ++j)
		{
			halfloat i_hf = __float2half(i);
			halfloat j_hf = __float2half(j);
			const int row = i + h_input;
			const int col = j + w_input;
			const int kernel_index = i * kernel_w + j;

			const int offset_h_offset = 2 * kernel_index * f_area_output + h_output * width_output + w_output;
			const int offset_w_offset = (2 * kernel_index + 1) * f_area_output + h_output * width_output + w_output;
			const int mask_offset = kernel_index * f_area_output + h_output * width_output + w_output;

			const halfloat offset_h = data_offset_ptr[offset_h_offset];
			const halfloat offset_w = data_offset_ptr[offset_w_offset];
			const halfloat mask = data_mask_ptr[mask_offset];

			halfloat val = zero;
			halfloat h_im = h_input_hf + i_hf * dilation_h_hf + offset_h;
			halfloat w_im = w_input_hf + j_hf * dilation_w_hf + offset_w;

			if (h_im > n_one && w_im > n_one && h_im < height_input_hf && w_im < width_input_hf)
			{
				val = dmcnIm2colBilinearFP16(data_input_ptr, width_input_hf, height_input_hf, width_input_hf, h_im, w_im);
			}
			*data_output_ptr = val * mask;
			data_output_ptr += f_area_output;
		}
	}
}

template<typename _T>
static __global__ void biasKernel(_T* data_input, const _T* bias, const int f_area, int edge) {

	KERNEL_POSITION;
	int bias_index = position / f_area;
	data_input[position] += bias[bias_index];
}

template<typename _T>
inline void segemm_native(hipblasHandle_t handle,
	hipblasOperation_t transa,
	hipblasOperation_t transb,
	int m,
	int n,
	int k,
	float alpha, /* host or device pointer */
	const _T *A,
	int lda,
	const _T *B,
	int ldb,
	float beta, /* host or device pointer */
	_T *C,
	int ldc);

template<>
inline void segemm_native<float>(hipblasHandle_t handle,
	hipblasOperation_t transa,
	hipblasOperation_t transb,
	int m,
	int n,
	int k,
	float alpha, /* host or device pointer */
	const float *A,
	int lda,
	const float *B,
	int ldb,
	float beta, /* host or device pointer */
	float *C,
	int ldc) {
	cublasCheck(hipblasSgemm(handle, transa, transb, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
	//cublasCheck(hipblasGemmEx(handle, transa, transb, m, n, k, &alpha, A, HIP_R_32F, lda, B, HIP_R_32F, ldb, &beta, C, HIP_R_32F, ldc, HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
}

template<>
inline void segemm_native<TRTInfer::halfloat>(hipblasHandle_t handle,
	hipblasOperation_t transa,
	hipblasOperation_t transb,
	int m,
	int n,
	int k,
	float alpha,
	const TRTInfer::halfloat *A,
	int lda,
	const TRTInfer::halfloat *B,
	int ldb,
	float beta, 
	TRTInfer::halfloat *C,
	int ldc) {

	auto halpha = TRTInfer::halfloat(alpha);
	auto hbeta = TRTInfer::halfloat(beta);
	//cublasCheck(hipblasHgemm(handle, transa, transb, m, n, k, &halpha, A, lda, B, ldb, &hbeta, C, ldc));
	cublasCheck(hipblasGemmEx(handle, transa, transb, m, n, k, &halpha, A, HIP_R_16F, lda, B, HIP_R_16F, ldb, &hbeta, C, HIP_R_16F, ldc, HIP_R_16F, HIPBLAS_GEMM_DEFAULT));
}

template<typename _T>
static void enqueue_native(hipblasHandle_t handle, const std::vector<GTensor>& inputs, std::vector<GTensor>& outputs, const std::vector<GTensor>& weights, void* workspace, hipStream_t stream) {
	auto& data = inputs[0];
	auto& om = inputs[1];
	auto& out = outputs[0];

	int kernel_size = weights[0].width_;
	int deformable_group = 1;
	size_t maskSize = (size_t)data.height_ * data.width_ * kernel_size * kernel_size * deformable_group;
	size_t im2colSize = (size_t)data.channel_ * kernel_size * kernel_size * out.height_ * out.width_;

	const int m = out.channel_;
	const int n = out.count(2);
	const int k = data.channel_ * kernel_size * kernel_size;
	float alpha = 1.0;
	float beta = 0.0;

	cublasCheck(hipblasSetStream(handle, stream));
	for (int ibatch = 0; ibatch < data.num_; ++ibatch) {
		_T* maskWorkspacePtr = (_T*)workspace + (maskSize + im2colSize) * ibatch;
		_T* im2colWorkspacePtr = (_T*)workspace + (maskSize + im2colSize) * ibatch + maskSize;

		_T* inputMask = om.ptr<_T>(ibatch, om.channel_ / 3 * 2);
		ExecuteKernel(maskSize, sigmoidKernel, stream)(inputMask, maskWorkspacePtr, maskSize);

		_T* datainput = data.ptr<_T>(ibatch);
		_T* offset = om.ptr<_T>(ibatch);

		ExecuteKernel(im2colSize, DCNIm2colKernel, stream)(
			datainput, offset, maskWorkspacePtr, data.height_, data.width_, kernel_size, kernel_size, 1, 1, 1, 1, 1, 1, data.channel_, data.num_, data.channel_, deformable_group,
			out.height_, out.width_, im2colWorkspacePtr, im2colSize);

		_T* weightKernel = weights[0].ptr<_T>();
		segemm_native(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, im2colWorkspacePtr, n, weightKernel, k, beta, out.ptr<_T>(ibatch), n);

		if (weights.size() > 1) {
			_T* weightBias = weights[1].ptr<_T>();
			size_t edge = out.count(1);
			size_t area = out.count(2);
			ExecuteKernel(edge, biasKernel, stream)(out.ptr<_T>(ibatch), weightBias, area, edge);
		}
	}
}

int DCNv2::initialize(){
	cublasCheck(hipblasCreate(&cublasHandle_));
	return 0;
}

void DCNv2::terminate(){
	cublasCheck(hipblasDestroy(cublasHandle_));
	cublasHandle_ = nullptr;
}

int DCNv2::enqueue(const std::vector<GTensor>& inputs, std::vector<GTensor>& outputs, const std::vector<GTensor>& weights, void* workspace, hipStream_t stream) {
	
	if (config_->configDataType_ == TRTInfer::DataType::dtFloat) {
		enqueue_native<float>(cublasHandle_, inputs, outputs, weights, workspace, stream);
	}
	else if (config_->configDataType_ == TRTInfer::DataType::dtHalfloat) {
		enqueue_native<TRTInfer::halfloat>(cublasHandle_, inputs, outputs, weights, workspace, stream);
	}
	return 0;
}

nvinfer1::Dims DCNv2::outputDims(int index, const nvinfer1::Dims* inputDims, int nbInputDims) {
	//INFO("inputDims = %d, %d, %d, %d, %d", inputDims[0].nbDims, inputDims[0].d[0], inputDims[0].d[1], inputDims[0].d[2], inputDims[0].d[3]);
	return nvinfer1::Dims3(config_->weights_[0]->num(), inputDims[0].d[1], inputDims[0].d[2]);
}

size_t DCNv2::getWorkspaceSize(int maxBatchSize) const {

	int kernel_size = 3;
	int deformable_group = 1;

	//inputChannel * k * k * outputHeight * outputWidth
	size_t im2colSize = (size_t)config_->input[0].d[0] * kernel_size * kernel_size * config_->output[0].d[1] * config_->output[0].d[2];
	size_t maskSize = (size_t)config_->input[0].d[1] * config_->input[0].d[2] * kernel_size * kernel_size * deformable_group;
	config_->workspaceSize_ = (im2colSize + maskSize) * maxBatchSize * TRTInfer::dataTypeSize(config_->configDataType_);
	return config_->workspaceSize_;
}

std::shared_ptr<LayerConfig> DCNv2::config(const std::string& layerName) {
	auto cfg = TRTPlugin::config(layerName);

	cfg->supportDataType_ = {nvinfer1::DataType::kFLOAT};
	//cfg->supportDataType_ = {nvinfer1::DataType::kHALF, nvinfer1::DataType::kFLOAT};
	//cfg->supportDataType_ = {nvinfer1::DataType::kHALF};
	return cfg;
}

RegisterPlugin(DCNv2);