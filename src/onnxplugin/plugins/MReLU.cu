#include "hip/hip_runtime.h"


#include "MReLU.hpp"
#include <json.hpp>

typedef TRTInfer::halfloat halfloat;

template<typename _T>
__global__ void MReLUKernel(_T* input, _T* output, _T bias, int edge);


template<>
__global__ void MReLUKernel(float* input, float* output, float bias, int edge) {

    KERNEL_POSITION;
    float x = input[position];
    float a = x > 0 ? x : 0;
	output[position] = a + bias;
}

template<>
__global__ void MReLUKernel(halfloat* input, halfloat* output, halfloat bias, int edge) {

	KERNEL_POSITION;

	halfloat x = input[position];
    halfloat _zero = 0.0f;
    x = x > _zero ? x : _zero;
	output[position] = x + bias;
}

void MReLUConfig::init(){
    INFO("init MReLU config: %s", info_.c_str());
	INFO("MReLU weights = %d[%s]", this->weights_.size(), this->weights_[0]->shapeString());
	
	Json::Value value;
	if(Json::Reader().parse(info_, value)){
		INFO("MReLU kernel_size: %d", value["kernel_size"].asInt());
		INFO("MReLU eps: %g", value["eps"].asFloat());
		INFO("MReLU other: %s", value["other"].asCString());
	}
}

nvinfer1::Dims MReLU::outputDims(int index, const nvinfer1::Dims* inputDims, int nbInputDims) {
	return inputDims[0];
}

std::shared_ptr<LayerConfig> MReLU::config(const std::string& layerName) {
	auto cfg = std::shared_ptr<LayerConfig>(new MReLUConfig());

	//定义我们这个插件支持half和float格式
	cfg->supportDataType_ = {nvinfer1::DataType::kHALF, nvinfer1::DataType::kFLOAT};
	//cfg->supportDataType_ = {nvinfer1::DataType::kHALF};
	return cfg;
}

int MReLU::enqueue(const std::vector<GTensor>& inputs, std::vector<GTensor>& outputs, const std::vector<GTensor>& weights, void* workspace, hipStream_t stream) {

	int count = inputs[0].count();
	auto grid = gridDims(count);
	auto block = blockDims(count);
	float bias = *this->config_->weights_[0]->cpu<float>();

	if (config_->configDataType_ == TRTInfer::DataType::dtFloat) {
		MReLUKernel <<<grid, block >>> (inputs[0].ptr<float>(), outputs[0].ptr<float>(), bias, count);
	}
	else if (config_->configDataType_ == TRTInfer::DataType::dtHalfloat) {
		MReLUKernel <<<grid, block>>> (inputs[0].ptr<halfloat>(), outputs[0].ptr<halfloat>(), halfloat(bias), count);
	}
	return 0;
}

RegisterPlugin(MReLU);