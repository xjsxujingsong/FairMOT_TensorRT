#include "hip/hip_runtime.h"


#include "HSwish.hpp"

typedef TRTInfer::halfloat halfloat;

template<typename _T>
__global__ void HSwishKernel(_T* input, _T* output, int edge);


template<>
__global__ void HSwishKernel(float* input, float* output, int edge) {

    KERNEL_POSITION;
    float x = input[position];
    float a = x + 3;
    a = a < 0 ? 0 : (a >= 6 ? 6 : a);
	output[position] = x * a / 6;
}

template<>
__global__ void HSwishKernel(halfloat* input, halfloat* output, int edge) {

	KERNEL_POSITION;

    halfloat _six = 6.0f;
	halfloat x = input[position];
    halfloat a = x + halfloat(3.0f);
    halfloat _zero = 0.0f;
    a = a < _zero ? _zero : (a >= _six ? _six : a);
	output[position] = x * a / _six;
}

void HSwishConfig::init(){
    //INFO("init hswish config: %s", info_.c_str());
    //INFO("weights = %d", this->weights_.size());
}

nvinfer1::Dims HSwish::outputDims(int index, const nvinfer1::Dims* inputDims, int nbInputDims) {
	return inputDims[0];
}

std::shared_ptr<LayerConfig> HSwish::config(const std::string& layerName) {
	auto cfg = std::shared_ptr<LayerConfig>(new HSwishConfig());

	//定义我们这个插件支持half和float格式
	cfg->supportDataType_ = {nvinfer1::DataType::kHALF, nvinfer1::DataType::kFLOAT};
	//cfg->supportDataType_ = {nvinfer1::DataType::kHALF};
	return cfg;
}

int HSwish::enqueue(const std::vector<GTensor>& inputs, std::vector<GTensor>& outputs, const std::vector<GTensor>& weights, void* workspace, hipStream_t stream) {

	int count = inputs[0].count();
	auto grid = gridDims(count);
	auto block = blockDims(count);

	if (config_->configDataType_ == TRTInfer::DataType::dtFloat) {
		HSwishKernel <<<grid, block >>> (inputs[0].ptr<float>(), outputs[0].ptr<float>(), count);
	}
	else if (config_->configDataType_ == TRTInfer::DataType::dtHalfloat) {
		HSwishKernel <<<grid, block>>> (inputs[0].ptr<halfloat>(), outputs[0].ptr<halfloat>(), count);
	}
	return 0;
}

RegisterPlugin(HSwish);